#include "hip/hip_runtime.h"
%%writefile add_vector_task.cu

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include <cmath>
#include <algorithm>
#include <string>

#define CUDA_CHECK(call)                                                     \
  do {                                                                       \
    hipError_t _e = (call);                                                 \
    if (_e != hipSuccess) {                                                 \
      fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,          \
              hipGetErrorString(_e));                                       \
      std::exit(EXIT_FAILURE);                                               \
    }                                                                        \
  } while (0)

__global__ void addVector(const float* __restrict__ A,
                       const float* __restrict__ B,
                       float* __restrict__ C,
                       size_t N)
{
    // grid-stride loop for any N
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * (size_t)gridDim.x;
    for (size_t i = idx; i < N; i += stride) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char** argv) {
    size_t N = (argc > 1) ? std::stoull(argv[1]) : 50'000'000ULL; // 50M by default
    size_t bytes = N * sizeof(float);
    printf("Vector addition N=%zu (%.2f MB per array)\n", N, bytes/1e6);

    int dev = 0;
    CUDA_CHECK(hipSetDevice(dev));
    hipDeviceProp_t prop{};
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev));
    printf("GPU: %s, SM %d.%d, globalMem=%.1f GB, memClock=%.0f MHz, memBusWidth=%d-bit\n",
           prop.name, prop.major, prop.minor,
           prop.totalGlobalMem / (1024.0*1024.0*1024.0),
           prop.memoryClockRate/1000.0, prop.memoryBusWidth);

    float *hA, *hB, *hC;
    CUDA_CHECK(hipHostMalloc(&hA, bytes));
    CUDA_CHECK(hipHostMalloc(&hB, bytes));
    CUDA_CHECK(hipHostMalloc(&hC, bytes));

    for (size_t i = 0; i < N; ++i) {
        hA[i] = 1.0f;             // simple pattern
        hB[i] = (float)(i % 100); // varies to avoid trivial constant folding
    }

    auto t0 = std::chrono::high_resolution_clock::now();
    float* hRef = (float*)malloc(bytes);
    for (size_t i = 0; i < N; ++i) hRef[i] = hA[i] + hB[i];
    auto t1 = std::chrono::high_resolution_clock::now();
    double cpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();

    float *dA, *dB, *dC;
    CUDA_CHECK(hipMalloc(&dA, bytes));
    CUDA_CHECK(hipMalloc(&dB, bytes));
    CUDA_CHECK(hipMalloc(&dC, bytes));

    hipEvent_t eStart, eAfterH2D, eAfterKernel, eStop;
    CUDA_CHECK(hipEventCreate(&eStart));
    CUDA_CHECK(hipEventCreate(&eAfterH2D));
    CUDA_CHECK(hipEventCreate(&eAfterKernel));
    CUDA_CHECK(hipEventCreate(&eStop));

    int block = 256;
    int smCount = prop.multiProcessorCount;
    // 4 blocks per SM is a decent start; tweak if you like
    int grid = std::min<int>((int)((N + block - 1) / block), smCount * 4);

    // --- Time Host2Device + Kernel + Device2Host end-to-end ---
    CUDA_CHECK(hipEventRecord(eStart));

    CUDA_CHECK(hipMemcpy(dA, hA, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, bytes, hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(eAfterH2D));

    addVector<<<grid, block>>>(dA, dB, dC, N);
    CUDA_CHECK(hipGetLastError());   // check kernel launch
    CUDA_CHECK(hipEventRecord(eAfterKernel));

    CUDA_CHECK(hipMemcpy(hC, dC, bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventRecord(eStop));
    CUDA_CHECK(hipEventSynchronize(eStop));

    float h2d_ms=0, kern_ms=0, d2h_ms=0, total_ms=0;
    CUDA_CHECK(hipEventElapsedTime(&total_ms, eStart, eStop));
    CUDA_CHECK(hipEventElapsedTime(&h2d_ms,  eStart, eAfterH2D));
    CUDA_CHECK(hipEventElapsedTime(&kern_ms, eAfterH2D, eAfterKernel));
    CUDA_CHECK(hipEventElapsedTime(&d2h_ms,  eAfterKernel, eStop));

    double max_abs_err = 0.0;
    for (size_t i = 0; i < N; ++i) {
        max_abs_err = std::max(max_abs_err, (double)std::abs(hRef[i] - hC[i]));
    }
    printf("Verification: max |CPU-GPU| = %.6g\n", max_abs_err);

    // Report
    printf("\nCPU (single-thread)              : %8.3f ms\n", cpu_ms);
    printf("GPU Host2Device                    : %8.3f ms\n", h2d_ms);
    printf("GPU Kernel                         : %8.3f ms\n", kern_ms);
    printf("GPU Device2Host                    : %8.3f ms\n", d2h_ms);
    printf("GPU Total (H2D + Kernel + D2H)     : %8.3f ms\n", total_ms);

    double bytes_moved = 3.0 * bytes; // A,B to device + C back
    double gb_moved = bytes_moved / 1e9;
    double gbps_e2e = gb_moved / (total_ms / 1e3);
    double gbps_kernel_bound = (3.0 * bytes) / (kern_ms / 1e3) / 1e9; // if mem-limited in device
    printf("\nApprox bandwidth (E2E including PCIe): %.2f GB/s\n", gbps_e2e);
    printf("Approx device bandwidth (kernel-only) : %.2f GB/s (rough upper bound)\n", gbps_kernel_bound);

    // Cleanup
    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dC));
    CUDA_CHECK(hipHostFree(hA));
    CUDA_CHECK(hipHostFree(hB));
    CUDA_CHECK(hipHostFree(hC));
    free(hRef);
    CUDA_CHECK(hipEventDestroy(eStart));
    CUDA_CHECK(hipEventDestroy(eAfterH2D));
    CUDA_CHECK(hipEventDestroy(eAfterKernel));
    CUDA_CHECK(hipEventDestroy(eStop));
    return 0;
}
